/*
This is by far the fastest time of all systems.

1000 bodies for 10 min
real	0m6.662s
user	0m4.099s
sys	0m2.546s


8000 bodies for 10 min
real	2m3.830s
user	1m16.900s
sys	0m46.798s
*/

#include <hip/hip_runtime.h>
#include<stdlib.h>
#include<stdio.h>
#include<math.h>

#define G 6.673e-11
#define MOON_MASS 7.34e22
#define MOON_RADIUS 1.7371e6

__global__
void update_velocity(int n, float *position, float *velocity, float *mass, float duration) {
  float r1, r2, r3, rsquared, accel, normal, dvx = 0, dvy = 0, dvz = 0;
  int j, i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < n) {
#define LOOP                                                        \
        /* compute the force of gravity */                          \
        r1 = (position[j*3 + 0] - position[i*3 + 0]);               \
        r2 = (position[j*3 + 1] - position[i*3 + 1]);               \
        r3 = (position[j*3 + 2] - position[i*3 + 2]);               \
        rsquared = r1*r1 + r2*r2 + r3*r3;                           \
        accel = (G * mass[j]) / rsquared;                           \
                                                                    \
        /* compute the normal vector pointing from i to j */        \
        normal = 1 / sqrtf(rsquared);                               \
                                                                    \
        /* now update the velocity */                               \
        dvx += r1 * normal * accel * duration;                      \
        dvy += r2 * normal * accel * duration;                      \
        dvz += r3 * normal * accel * duration;                      \

    for(j = 0; j < i; j++) {
      LOOP
    }
    for(j = i + 1; j < n; j++) {
      LOOP
    }

    velocity[i*3 + 0] += dvx;
    velocity[i*3 + 1] += dvy;
    velocity[i*3 + 2] += dvz;
  }
}

__global__
void update_position(int n, float *position, float *velocity, float duration) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i < n) {
    position[i*3 + 0] += velocity[i*3 + 0] * duration;
    position[i*3 + 1] += velocity[i*3 + 1] * duration;
    position[i*3 + 2] += velocity[i*3 + 2] * duration;
  }
}

/* This simulates a 1000 Kg body orbiting the moon at at distance of about 10 kilometers */
void two_body_test() {
  int i, n = 2;
  float position[6] = 
    { 0     , 0, 0, 
      1.75e6, 0, 0 };
  float velocity[6] = 
    { 0, 0      , 0,
      0, 1.673e3, 0 };
  float mass[2] = {MOON_MASS, 1e3};
  float *position_d, *velocity_d, *mass_d;
  float alt = 0;

  hipMalloc(&position_d, 6*sizeof(float));
  hipMalloc(&velocity_d, 6*sizeof(float));
  hipMalloc(&mass_d, 2*sizeof(float));
  hipMemcpy(position_d, position, 6*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(velocity_d, velocity, 6*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(mass_d, mass, 2*sizeof(float), hipMemcpyHostToDevice);

  while(1) {
    for(i = 0; i < 6000; i++) {
      update_velocity<<<2, 2>>>(n, position_d, velocity_d, mass_d, 0.1);
      update_position<<<2, 2>>>(n, position_d, velocity_d, 0.1);
    }  

    hipMemcpy(&position, position_d, 6*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&velocity, velocity_d, 6*sizeof(float), hipMemcpyDeviceToHost);
    alt = sqrt(powf((position[0*3 + 0] - position[1*3 + 0]), 2) +
               powf((position[0*3 + 1] - position[1*3 + 1]), 2) +
               powf((position[0*3 + 2] - position[1*3 + 2]), 2));
    printf("alt %f v %f,%f,%f\n", alt - MOON_RADIUS, 
           velocity[3+0], velocity[3+1], velocity[3+2]);
  }
}

float random_float() {
  float n = (float) rand();
  return (n / (log10f(n) + 1)); /* between 0 and 1 */
}

void many_body_test() {
  int i, n = 8000;
  float position[n*3];
  float velocity[n*3];
  float mass[n];
  float *position_d, *velocity_d, *mass_d;
  float alt;

  hipMalloc(&position_d, n*3*sizeof(float));
  hipMalloc(&velocity_d, n*3*sizeof(float));
  hipMalloc(&mass_d, n*sizeof(float));

  srand(1232);
  for(i = 0; i < n; i++) {
    position[i*3 + 0] = random_float() * 1e9;
    position[i*3 + 1] = random_float() * 1e9;
    position[i*3 + 2] = random_float() * 1e9;

    velocity[i*3 + 0] = random_float() * 5e2;
    velocity[i*3 + 1] = random_float() * 5e2;
    velocity[i*3 + 2] = random_float() * 5e2;

    mass[i] = random_float() * 1e22;
  }

  hipMemcpy(position_d, position, n*3*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(velocity_d, velocity, n*3*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(mass_d, mass, n*sizeof(float), hipMemcpyHostToDevice);

  #define B 16
  for(i = 0; i < 6000; i++) {
    update_velocity<<<(n+(B-1))/B, B>>>(n, position_d, velocity_d, mass_d, 0.1);
    update_position<<<(n+(B-1))/B, B>>>(n, position_d, velocity_d, 0.1);
  }

  hipMemcpy(&position, position_d, n*3*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&velocity, velocity_d, n*3*sizeof(float), hipMemcpyDeviceToHost);
  alt = sqrt(powf((position[(n-1)*3 + 0] - position[1*3 + 0]), 2) +
             powf((position[(n-1)*3 + 1] - position[1*3 + 1]), 2) +
             powf((position[(n-1)*3 + 2] - position[1*3 + 2]), 2));

  printf("alt %f v %f,%f,%f\n", alt, 
         velocity[3+0], velocity[3+1], velocity[3+2]);
}

int main() {
  many_body_test();
  return 0;
}
